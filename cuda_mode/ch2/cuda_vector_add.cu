// this is a modification of the vector_add.c with cuda kernel implemented
#include <stdio.h>
#include <hip/hip_runtime.h>

// compute vector sum c = a + b
// each thread performs one pair-wise addition
// calling this kernel launches a new grid of threads
__global__
void vecAddKernel(float* A, float* B, float* C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x; // very important to understand blockDim vs blockIdx
    if (i < n){
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n){
    int size = n * sizeof(float); // size of a float data type * the length of the array
    float *A_d, *B_d, *C_d;

    // part 1: allocate device memory for A, B and C
    // copy A and B in device memory
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // part 2: call kernel - to launch a grid of kernels
    // to perform the actual vector addition
    // <<<dimGrid, dimBlock>>>
    vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);
    
    // part 3: copy c from the device memory
    // free device vectors
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main()
{
    float A[5] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float B[5] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float C[5];

    vecAdd(A, B, C, 5);

    return 0;
}