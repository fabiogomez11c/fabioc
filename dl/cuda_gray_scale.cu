#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void colorToGrayscaleConvertion(
        unsigned char * Pout,
        unsigned char * Pin,
        int width,
        int height
    )
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height){
        // get 1d offset for the grayscale image
        int grayOffset = row * width + col;

        int rgbOffset = grayOffset*CHANNELS;
        unsigned char r = Pin[rgbOffset    ]; // red
        unsigned char g = Pin[rgbOffset + 1]; // green
        unsigned char b = Pin[rgbOffset + 2]; // blue

        Pout[grayOffset] = 0.21f*r + 0.71f*b + 0.07f*b; // operations with float are faster than double
    }

}

int main()
{
    int m;
    int n;
    dim3 dimGrid(ceil(m/16.0), ceil(m/16.0), 1);
    dim3 dimBlock(16, 16, 1);

    // class kernel
    colorToGrayscaleConvertion<<<dimGrid, dimBlock>>>
        (Pin_d, Pout_d, m, n)

    return 0;
}