// this is a modification of the vector_add.c with cuda kernel implemented
#include <stdio.h>
#include <hip/hip_runtime.h>

// compute vector sum c = a + b
// each thread performs one pair-wise addition
// calling this kernel launches a new grid of threads
__global__
void vecAddKernel(float* A, float* B, float* C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n){
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n){
    int size = n * sizeof(float); // size of a float data type * the length of the array
    float *A_d, *B_d, *C_d;

    // part 1: allocate device memory for A, B and C
    // copy A and B in device memory
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // part 2: call kernel - to launch a grid of kernels
    // to perform the actual vector addition
    
    // part 3: copy c from the device memory
    // free device vectors
    hipMemcpy(C_d, C_h, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main()
{
    return 0;
}